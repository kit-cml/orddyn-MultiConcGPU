#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "modules/cipa_t.cuh"
#include "modules/drug_conc.hpp"
#include "modules/glob_funct.hpp"
#include "modules/glob_type.hpp"
#include "modules/gpu.cuh"

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <filesystem>
#include <iostream>
#include <math.h>
#include <regex>
#include <string>
#include <sys/stat.h>
#include <unordered_map>
#include <vector>
namespace fs = std::filesystem;

#define ENOUGH ((CHAR_BIT * sizeof(int) - 1) / 3 + 3)
char buffer[255];

const unsigned int datapoint_size = 7500;
const unsigned int sample_limit = 10000;

// Custom comparison function for C-style strings
struct CStringCompare {
    bool operator()(const char *lhs, const char *rhs) const {
        return std::strcmp(lhs, rhs) < 0;
    }
};

// Function to get unique C-style strings from an array
// const char** getUniqueStrings(const char** stringArray, size_t arraySize, size_t& uniqueSize) {
//     // Use std::set with custom comparison function to get unique values
//     std::set<const char*, CStringCompare> uniqueStringsSet(stringArray, stringArray + arraySize);

//     // Create a new array to store unique strings
//     const char** uniqueArray = new const char*[uniqueStringsSet.size()];
//     size_t index = 0;

//     // Copy unique strings to the new array
//     for (const auto& element : uniqueStringsSet) {
//         uniqueArray[index++] = element;
//     }

//     // Update the output parameter with the size of the unique array
//     uniqueSize = uniqueStringsSet.size();

//     return uniqueArray;
// }

// Function to create a bag of words with insertion order from an array of C-style strings
std::map<std::string, int> createBagOfWords(char *textArray[], size_t arraySize, std::vector<std::string> &insertionOrder) {
    std::map<std::string, int> bagOfWords;

    for (size_t i = 0; i < arraySize; ++i) {
        // Copy the string to a mutable buffer
        char buffer[32]; // Adjust the size as needed
        strcpy(buffer, textArray[i]);
        insertionOrder.push_back(buffer);
    }

    return bagOfWords;
}

clock_t START_TIMER;

clock_t tic();
void toc(clock_t start = START_TIMER);

clock_t tic() {
    return START_TIMER = clock();
}

void toc(clock_t start) {
    std::cout
        << "Elapsed time: "
        << (clock() - start) / (double)CLOCKS_PER_SEC << "s"
        << std::endl;
}

void addDrugData(char*** arrayOfStrings, int& size, const char newString[]) {
    // Allocate memory for a new array with increased size
    char** newArray = new char*[size + 1];

    // Copy existing strings to the new array
    for (int i = 0; i < size; ++i) {
        newArray[i] = new char[strlen((*arrayOfStrings)[i]) + 1];
        strcpy(newArray[i], (*arrayOfStrings)[i]);
        delete[] (*arrayOfStrings)[i]; // Deallocate memory for old strings
    }

    // Allocate memory for the new string and copy it
    newArray[size] = new char[strlen(newString) + 1];
    strcpy(newArray[size], newString);

    // Deallocate memory for the old array
    delete[] *arrayOfStrings;

    // Update the pointer to point to the new array
    *arrayOfStrings = newArray;

    // Increment the size
    ++size;
}

void prepingGPUMemory(double *&d_ALGEBRAIC, int num_of_algebraic, int sample_size, double *&d_CONSTANTS, int num_of_constants, double *&d_RATES, int num_of_rates, double *&d_STATES, int num_of_states, param_t *&d_p_param, cipa_t *&temp_result, cipa_t *&cipa_result, double *&d_STATES_RESULT, double *&d_ic50, double *ic50, double *&d_conc, double *conc, double *&d_herg, double *herg, param_t *p_param) {
    printf("preparing GPU memory space \n");
    hipMalloc(&d_ALGEBRAIC, num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_CONSTANTS, num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_RATES, num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_STATES, num_of_states * sample_size * sizeof(double));

    hipMalloc(&d_p_param, sizeof(param_t));

    // prep for 1 cycle plus a bit (7000 * sample_size)
    hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
    hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));

    hipMalloc(&d_STATES_RESULT, num_of_states * sample_size * sizeof(double));
    
    hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
    // hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));
    hipMalloc(&d_conc, sample_size * sizeof(double));
    hipMalloc(&d_herg, 6 * sizeof(double));

    printf("Copying sample files to GPU memory space \n");
    hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_herg, herg, 6 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_conc, conc, sample_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);
}

void freeingGPUMemory(double *d_ALGEBRAIC, double *d_CONSTANTS, double *d_RATES, double *d_STATES, param_t *d_p_param, cipa_t *temp_result, cipa_t *cipa_result, double *d_STATES_RESULT, double *d_ic50, double *d_herg) {
    hipFree(d_ALGEBRAIC);
    hipFree(d_CONSTANTS);
    hipFree(d_RATES);
    hipFree(d_STATES);
    hipFree(d_p_param);
    hipFree(temp_result);
    hipFree(cipa_result);
    hipFree(d_STATES_RESULT);
    hipFree(d_ic50);
    hipFree(d_herg);
}

int gpu_check(unsigned int datasize) {
    int num_gpus;
    float percent;
    int id;
    size_t free, total;
    hipGetDeviceCount(&num_gpus);
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        hipSetDevice(gpu_id);
        hipGetDevice(&id);
        hipMemGetInfo(&free, &total);
        percent = (free / (float)total);
        printf("GPU No %d\nFree Memory: %ld, Total Memory: %ld (%f percent free)\n", id, free, total, percent * 100.0);
    }
    percent = 1.0 - (datasize / (float)total);

    return 0;
}

int get_IC50_data_from_file(const char *file_name, double *ic50) {
    /*
    a host function to take all samples from the file, assuming each sample has 14 features.

    it takes the file name, and an ic50 (already declared in 1D, everything become 1D)
    as a note, the data will be stored in 1D array, means this functions applies flatten.

    it returns 'how many samples were detected?' in integer.
    */
    FILE *fp_drugs;
    //   drug_t ic50;
    char *token;
    char buffer_ic50[255];
    unsigned int idx;

    if ((fp_drugs = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n",
               file_name);
        return 0;
    }
    idx = 0;
    int sample_size = 0;
    fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs);                  // skip header
    while (fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs) != NULL) { // begin line reading
        token = strtok(buffer_ic50, ",");
        while (token != NULL) { // begin data tokenizing
            ic50[idx++] = strtod(token, NULL);
            token = strtok(NULL, ",");
        } // end data tokenizing
        sample_size++;
    } // end line reading

    fclose(fp_drugs);
    return sample_size;
}

// TODO: NewFile 3. Create new function that takes several params
int get_IC50_data_from_file(const char *file_name, double *ic50, double *conc, char **drug_name) {
    /*
    a host function to take all samples from the file, assuming each sample has 14 features.

    it takes the file name, and an ic50 (already declared in 1D, everything become 1D)
    as a note, the data will be stored in 1D array, means this functions applies flatten.

    it returns 'how many samples were detected?' in integer.
    */
    FILE *fp_drugs;
    //   drug_t ic50;
    char *token;
    char tmp_drug_name[32];
    char buffer_ic50[255];
    unsigned int idx_ic50, idx_conc;
    int drugsize = 0;

    if ((fp_drugs = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n",
               file_name);
        return 0;
    }
    idx_ic50 = 0;
    idx_conc = 0;
    int sample_size = 0;
    fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs);                  // skip header
    while (fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs) != NULL) { // begin line reading
        /*
        TODO: Extracting token from file
        1. take token for each file
        2. check the first token to drug_name, if already exist in array, then skip it
        3. check the second token to conc
        */

        token = strtok(buffer_ic50, ",");
        printf("%s\n", token); // testingAuto
        strcpy(tmp_drug_name, token);
        token = strtok(NULL, ",");
        printf("%s\n", token); // testingAuto
        strcat(tmp_drug_name, "_");
        strcat(tmp_drug_name, token);
        
        printf("%s\n", tmp_drug_name); // testingAuto
        addDrugData(&drug_name, drugsize, tmp_drug_name);
        conc[idx_conc++] = strtod(token, NULL);
        token = strtok(NULL, ",");
        // Check if there is wrong in here
        while (token != NULL) { // begin data tokenizing
            ic50[idx_ic50++] = strtod(token, NULL);
            printf("%s\n", token); // testingAuto
            token = strtok(NULL, ",");
        } // end data tokenizing
        sample_size++;
    } // end line reading

    fclose(fp_drugs);
    return sample_size;
}

int get_cvar_data_from_file(const char *file_name, unsigned int limit, double *cvar) {
    // buffer for writing in snprintf() function
    char buffer_cvar[255];
    FILE *fp_cvar;
    // cvar_t cvar;
    char *token;
    // std::array<double,18> temp_array;
    unsigned int idx;

    if ((fp_cvar = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n",
               file_name);
    }
    idx = 0;
    int sample_size = 0;
    fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar);                                             // skip header
    while ((fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar) != NULL) && (sample_size < limit)) { // begin line reading
        token = strtok(buffer_cvar, ",");
        while (token != NULL) { // begin data tokenizing
            cvar[idx++] = strtod(token, NULL);
            // printf("%lf\n",cvar[idx]);
            token = strtok(NULL, ",");
        } // end data tokenizing
        // printf("\n");
        sample_size++;
        // cvar.push_back(temp_array);
    } // end line reading

    fclose(fp_cvar);
    return sample_size;
}

drug_t get_IC50_data_from_file(const char *file_name);
// return error and message based on the IC50 data
int check_IC50_content(const drug_t *ic50, const param_t *p_param) {
    if (ic50->size() == 0) {
        printf("Something problem with the IC50 file!\n");
        return 1;
    } else if (ic50->size() > 2000) {
        printf("Too much input! Maximum sample data is 2000!\n");
        return 2;
    } else if (p_param->pace_max < 750 && p_param->pace_max > 1000) {
        printf("Make sure the maximum pace is around 750 to 1000!\n");
        return 3;
    } else {
        return 0;
    }
}

int get_herg_data_from_file(const char* dir_name, char* drugname, double *herg)
{
  FILE *fp_herg;
  char *token;
  char full_herg_file_name[150];
  char buffer_herg[255];
  unsigned int idx;

  strcpy(full_herg_file_name, dir_name);
  strcat(full_herg_file_name,"/");
  strcat(drugname,".csv");
  strcat(full_herg_file_name,drugname);

  printf("reading herg file: %s\n",full_herg_file_name);

  if( (fp_herg = fopen(full_herg_file_name, "r")) == NULL){
    printf("Cannot open file %s\n", full_herg_file_name);
    return 0;
  }
  idx = 0;
  int sample_size = 0;
  fgets(buffer_herg, sizeof(buffer_herg), fp_herg); // skip header
  while( fgets(buffer_herg, sizeof(buffer_herg), fp_herg) != NULL )
    { // begin line reading
      token = strtok( buffer_herg, "," );
      while( token != NULL )
      { // begin data tokenizing
        herg[idx++] = strtod(token, NULL);
        token = strtok(NULL, ",");
      } // end data tokenizing
      sample_size++;
    } // end line reading

  fclose(fp_herg);
  printf("%lf, %lf, %lf, %lf, %lf, %lf\n",herg[0],herg[1],herg[2],herg[3],herg[4],herg[5]);
  return sample_size;
}

int main(int argc, char **argv) {
    /* TODO: Creating new init state that takes new file format
     * 1. Set the mechanism to iterate over file inside folder
     * 2. Take value of the csv file and put it inside pointer
     * 3. Create a filename based on first and second column similarity
     * 4.
     *
     */
    // enable real-time output in stdout
    setvbuf(stdout, NULL, _IONBF, 0);

    // NEW CODE STARTS HERE //
    // mycuda *thread_id;
    // hipMalloc(&thread_id, sizeof(mycuda));

    // input variables for cell simulation
    param_t *t_param;
    t_param = new param_t();
    t_param->init();
    edison_assign_params(argc, argv, t_param);
    char drug_dir[1024];
    strcpy(drug_dir, t_param->hill_file);

    // TODO: Automation 3. check file inside folder
    for (const auto &entry : fs::directory_iterator(drug_dir)) {
        param_t *p_param, *d_p_param;
        p_param = new param_t();
        p_param->init();
        edison_assign_params(argc, argv, p_param);

        std::filesystem::directory_entry dir_entry = entry;
        std::string entry_str = dir_entry.path().string();
        std::cout << entry_str << std::endl;
        std::regex pattern("/([a-zA-Z0-9_\.]+)\.csv");
        std::smatch match;
        std::regex_search(entry_str, match, pattern);

        // TODO: Automation 2. create drug_name and conc

        // TODO: NewFile 2. disable drug name for now since the file name is inside it
        // strcpy(p_param->drug_name, match[1].str().c_str());
        strcpy(p_param->hill_file, entry_str.c_str());
        strcpy(p_param->hill_file, entry_str.c_str());
        // strcat(p_param->hill_file, ".csv");
        // strcat(p_param->hill_file, "/IC50_samples.csv");

        // TODO: NewFile 3. getvalue from source is unnecessary
        // p_param->conc = getValue(drugConcentration, match[1].str()) * cmax;
        // p_param->show_val();

        double *ic50; // temporary
        double *cvar;
        double *conc;
        double *herg;
        char **drug_name = nullptr;

        ic50 = (double *)malloc(14 * sample_limit * sizeof(double));
        conc = (double *)malloc(sample_limit * sizeof(double));
        herg = (double *)malloc(6 * sizeof(double));

        double *d_ic50;
        double *d_conc;
        double *d_cvar;
        double *d_herg;
        double *d_ALGEBRAIC;
        double *d_CONSTANTS;
        double *d_RATES;
        double *d_STATES;
        double *d_STATES_RESULT;

        cipa_t *temp_result, *cipa_result;

        int num_of_constants = 146;
        int num_of_states = 41;
        int num_of_algebraic = 199;
        int num_of_rates = 41;

        printf("%s\n", p_param->hill_file); // testingAuto
        int sample_size = get_IC50_data_from_file(p_param->hill_file, ic50, conc, drug_name);
        int herg_size = get_herg_data_from_file(p_param->herg_dir, drug_name, herg);
        if (sample_size == 0)
            printf("Something problem with the IC50 file!\n");
        // else if(sample_size > 2000)
        //     printf("Too much input! Maximum sample data is 2000!\n");
        printf("Sample size: %d\n", sample_size);
        printf("Set GPU Number: %d\n", p_param->gpu_index);

        hipSetDevice(p_param->gpu_index);

        if (p_param->is_cvar == true) {
            int cvar_sample = get_cvar_data_from_file(p_param->cvar_file, sample_size, cvar);
            printf("Reading: %d Conductance Variability samples\n", cvar_sample);
        }

        prepingGPUMemory(d_ALGEBRAIC, num_of_algebraic, sample_size, d_CONSTANTS, num_of_constants, d_RATES, num_of_rates, d_STATES, num_of_states, d_p_param, temp_result, cipa_result, d_STATES_RESULT, d_ic50, ic50, d_conc, conc, d_herg, herg, p_param);

        tic();
        printf("Timer started, doing simulation.... \n\n\nGPU Usage at this moment: \n");
        const int thread = 32;
        int block = (sample_size + thread - 1) / thread;
        // int block = (sample_size + thread - 1) / thread;
        if (gpu_check(15 * sample_size * datapoint_size * sizeof(double) + sizeof(param_t)) == 1) {
            printf("GPU memory insufficient!\n");
            return 0;
        }
        printf("Sample size: %d\n", sample_size);
        hipSetDevice(p_param->gpu_index);
        printf("\n   Configuration: \n\n\tblock\t||\tthread\n---------------------------------------\n  \t%d\t||\t%d\n\n\n", block, thread);
        // initscr();
        // printf("[____________________________________________________________________________________________________]  0.00 %% \n");

        kernel_DrugSimulation<<<block, thread>>>(d_ic50, d_cvar, d_conc, d_herg, d_CONSTANTS, d_STATES, d_RATES, d_ALGEBRAIC,
                                                 d_STATES_RESULT,
                                                 sample_size,
                                                 temp_result, cipa_result,
                                                 d_p_param);
        // block per grid, threads per block
        // endwin();

        hipDeviceSynchronize();

        printf("allocating memory for computation result in the CPU, malloc style \n");
        double *h_states;
        cipa_t *h_cipa_result;

        h_states = (double *)malloc(num_of_states * sample_size * sizeof(double));
        h_cipa_result = (cipa_t *)malloc(sample_size * sizeof(cipa_t));
        printf("...allocating for all states, all set!\n");

        ////// copy the data back to CPU, and write them into file ////////
        printf("copying the data back to the CPU \n");

        hipMemcpy(h_states, d_STATES_RESULT, sample_size * num_of_states * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_cipa_result, cipa_result, sample_size * sizeof(cipa_t), hipMemcpyDeviceToHost);
        printf("Successfully reach here!!");
        // TODO: Automation 4. Free up GPU memory
        freeingGPUMemory(d_ALGEBRAIC, d_CONSTANTS, d_RATES, d_STATES,
                         d_p_param, temp_result, cipa_result, d_STATES_RESULT, d_ic50);

        FILE *writer;
        int check;
        bool folder_created = false;

        // TODO: writing to several files
        printf("writing to file... \n");
        char filename[500] = "./result/init_";
        char dvmdt_file[500];
        strcat(filename, match[1].str().c_str());
        strcat(filename, "/");
        if (folder_created == false) {
            check = mkdir(filename, 0777);
            // check if directory is created or not
            if (!check) {
                printf("Directory created\n");
            } else {
                printf("Unable to create directory\n");
            }
            folder_created = true;
        }

        // strcat(filename,conc_str);
        strcpy(dvmdt_file, filename);
        strcat(filename, "_state_only.csv");
        // sample loop
        writer = fopen(filename, "w");
        fprintf(writer, "V,CaMKt,cass,nai,nass,ki,kss,cansr,cajsr,cai,m,hf,hs,j,hsp,jp,mL,hL,hLp,a,iF,iS,ap,iFp,iSp,d,ff,fs,fcaf,fcas,jca,ffp,fcafp,nca,xrf,xrs,xs1,xs2,xk1,Jrelnp,Jrelp,\n");
        for (int sample_id = 0; sample_id < sample_size; sample_id++) {

            // fprintf(writer,"%d,",sample_id);
            for (int datapoint = 0; datapoint < num_of_states - 1; datapoint++) {
                // if (h_time[ sample_id + (datapoint * sample_size)] == 0.0) {continue;}
                fprintf(writer, "%lf,", // change this into string, or limit the decimal accuracy, so we can decrease filesize
                        h_states[(sample_id * num_of_states) + datapoint]);
            }
            fprintf(writer, "%lf\n", // write last data
                    h_states[(sample_id * num_of_states) + num_of_states - 1]

                    // 22.00
            );
        }
        fclose(writer);

        // dvmdt file
        strcat(dvmdt_file, "_dvmdt.csv");
        writer = fopen(dvmdt_file, "w");
        fprintf(writer, "Sample,dVm/dt\n");
        for (int sample_id = 0; sample_id < sample_size; sample_id++) {

            fprintf(writer, "%d,%lf\n", // write last data
                    sample_id,
                    h_cipa_result[sample_id].dvmdt_repol);
        }
        fclose(writer);

        freeingGPUMemory(d_ALGEBRAIC, d_CONSTANTS, d_RATES, d_STATES,
                         d_p_param, temp_result, cipa_result, d_STATES_RESULT, d_ic50, d_herg);

        free(h_states); free(h_cipa_result); 

        toc();
    }
    return 0;
}